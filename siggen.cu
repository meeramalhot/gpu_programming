#include "hip/hip_runtime.h"
// three matrixes, two rows in each matrix, three (columns) elements in each row
// int arr1[3][2][3] = {
//     {
//         {1, 2, 3},
//         {4, 5, 6}
//     },
//     {
//         {7, 8, 9},
//         {10, 11, 12}
//     },
//     {
// 		{13, 14, 15},
// 		{16, 17, 18}
//     }
// };


// int arr2[3][2][3] = {
//     {
//         {1, 2, 3},
//         {4, 5, 6}
//     },
//     {
//         {7, 8, 9},
//         {10, 11, 12}
//     },
//     {
// 		{13, 14, 15},
// 		{16, 17, 18}
//     }
// };


#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <inttypes.h>
#include <time.h>


__global__ void f_siggen(float* A, float* B, float* C, int rows, int cols) {
    
}

int main(int argc, char *argv[]) {
    if (argc != 3) {
        fprintf(stderr, "Usage: %s <rows> <cols>\n", argv[0]);
        return 1;
    }

    int rows = atoi(argv[1]);
    int cols = atoi(argv[2]);
    int size = rows * cols * sizeof(float);

    // host
    float* h_A;
    float* h_B;
    float* h_C;

    // device
    float* d_A;
    float* d_B;
    float* d_C;

    // allocate for host
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);
    
    // allocate on device
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    //    for (int i = 0; i < rows * cols; ++i) {

    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            h_A[i,j]=(float) ((i+j)%100)/2.0;
            h_B[i,j]=(float) 3.25*((i+j)%100);
        }
    }

    // hipMemcpy(dest,src,hipMemcpyDeviceToHost);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // ADD PARAMS
    dim3 threadsPerBlocks();
    dim3 numBlocks();

    f_siggen<<<block, threadsPerBlocks>>(d_A, d_B, D_C, rows, cols);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    //cleanup deivce
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipDeviceReset();

    //cleanup host
    free(h_A);
    free(h_B);
    free(h_C);
    
}