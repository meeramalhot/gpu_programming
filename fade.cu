#include "hip/hip_runtime.h"
__global__ void fade(unsigned char *d_in, unsigned char *d_out, float f, int xmax, int ymax) {

unsigned int idx, v;
int x = blockDim.x * blockIdx.x + threadIdx.x;
int y = blockDim.y * blockIdx.y + threatIdx.y;

if ((x >= xmax) || (y >= ymax)) {
    return;
}

idx = y * xmax + x;
v = d_in[idx] * f;
if (v>255) {
    v=255;
}
d_out[idx]=v;

}

int main(void) {
    dim3 nblocks(7,3);
    dim3 nthreads(16, 16);
    fade <<<nblocks, nthreads>>>(d_in, d_out, f, xmax, ymax); 
    hipDeviceSynchronize();
    hipDeviceReset();
}