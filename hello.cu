
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void) {
    printf("Hello from GPU\n");
}

int main(void) {
    printf("Hello from CPU\n");
    helloFromGPU <<< 2,10 >>>();
    hipDeviceSynchronize();
    hipDeviceReset();
}